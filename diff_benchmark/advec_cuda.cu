
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdio>
#include <chrono>
#include <cmath>

void init(
        float* const __restrict__ a,
        float* const __restrict__ u,
        float* const __restrict__ v,
        float* const __restrict__ w,
        float* const __restrict__ at,
        const int ncells)
{
    for (int i=0; i<ncells; ++i)
    {
        a[i] = pow(i,2)/pow(i+1,2);
        u[i] = pow(i,2)/pow(i+1,2);
        v[i] = pow(i,2)/pow(i+1,2);
        w[i] = pow(i,2)/pow(i+1,2);
        at[i] = 0.f;
    }
}


__device__ float interp2(const float a, const float b) { return 0.5f*(a + b); }


__global__ void diff(
        float* const __restrict__ at, const float* const __restrict__ a,
        const float* const __restrict__ u, const float* const __restrict__ v, const float* const __restrict__ w,
        const float dxi, const float dyi, const float dzi, 
        const int itot, const int jtot, const int ktot)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + 1;
    const int k = blockIdx.z + 1;

    const int ii = 1;
    const int jj = itot;
    const int kk = itot*jtot;

    if (i < itot-1 && j < jtot-1 && k < ktot-1)
    {
        const int ijk = i + j*jj + k*kk;
        at[ijk] +=
            - (  u[ijk+ii] * interp2(a[ijk   ], a[ijk+ii])
               - u[ijk   ] * interp2(a[ijk-ii], a[ijk   ]) ) * dxi

            - (  v[ijk+jj] * interp2(a[ijk   ], a[ijk+jj])
               - v[ijk   ] * interp2(a[ijk-jj], a[ijk   ]) ) * dyi

            - (  w[ijk+kk] * interp2(a[ijk   ], a[ijk+kk])
               - w[ijk   ] * interp2(a[ijk-kk], a[ijk   ]) ) * dzi;
    }
}

int main(int argc, char* argv[])
{
    if (argc != 2)
    {
        std::cout << "Add the grid size as an argument!" << std::endl;
        return 1;
    }

    const int nloop = 30;

    const int itot = std::stoi(argv[1]);
    const int jtot = std::stoi(argv[1]);
    const int ktot = std::stoi(argv[1]);
    const int ncells = itot*jtot*ktot;

    float *a = new float[ncells];
    float *u = new float[ncells];
    float *v = new float[ncells];
    float *w = new float[ncells];
    float *at = new float[ncells];
   
    init(a, u, v, w, at, ncells);

    float *a_cuda;
    float *u_cuda;
    float *v_cuda;
    float *w_cuda;
    float *at_cuda;

    hipMalloc(&a_cuda, ncells*sizeof(float));
    hipMalloc(&u_cuda, ncells*sizeof(float));
    hipMalloc(&v_cuda, ncells*sizeof(float));
    hipMalloc(&w_cuda, ncells*sizeof(float));
    hipMalloc(&at_cuda, ncells*sizeof(float));

    hipMemcpy(a_cuda, a, ncells*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(u_cuda, u, ncells*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v_cuda, v, ncells*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(w_cuda, w, ncells*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(at_cuda, at, ncells*sizeof(float), hipMemcpyHostToDevice);

    const int blocki = 64;
    const int blockj = 2;
    const int gridi = (itot-2)/blocki + ((itot-2)%blocki > 0);
    const int gridj = (jtot-2)/blockj + ((jtot-2)%blockj > 0);

    dim3 grid_gpu(gridi, gridj, ktot);
    dim3 block_gpu(blocki, blockj, 1);

    // Check results
    diff<<<grid_gpu, block_gpu>>>(
            at_cuda, a_cuda,
            u_cuda, v_cuda, w_cuda,
            0.1, 0.1, 0.1,
            itot, jtot, ktot);
 
    hipMemcpy(at, at_cuda, ncells*sizeof(float), hipMemcpyDeviceToHost);

    printf("at=%.20f\n",at[itot*jtot+itot+itot/2]);
 
    // Time performance 
    auto start = std::chrono::high_resolution_clock::now();

    for (int i=0; i<nloop; ++i)
        diff<<<grid_gpu, block_gpu>>>(
                at_cuda, a_cuda,
                u_cuda, v_cuda, w_cuda,
                0.1, 0.1, 0.1,
                itot, jtot, ktot);
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    double duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start).count();

    printf("time/iter = %E s (%i iters)\n",duration/(double)nloop, nloop);

    hipMemcpy(at, at_cuda, ncells*sizeof(float), hipMemcpyDeviceToHost);

    printf("at=%.20f\n", at[itot*jtot+itot+itot/4]);

    return 0;
}
